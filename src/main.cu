
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <map>

#include "omp.h"
#include <vector>

#include "../include/Logger.cuh"
#include "../include/FIleReader.cuh"
#include "../include/CGArray.cuh"

#include "../include/CSRCOO.cuh"

#include "../include/main_support.cuh"

#include "../include/Config.h"
#include "../include/ScanLarge.cuh"

#include "../subgraph_matching/subgraph_matching.cuh"

using namespace std;
//#define TriListConstruct

int main(int argc, char **argv)
{

    // CUDA_RUNTIME(hipDeviceReset());
    Config config = parseArgs(argc, argv);
    setbuf(stdout, NULL);
    printf("\033[0m");
    printf("Welcome ---------------------\n");
    printConfig(config);

    graph::MtB_Writer mwriter;
    auto fileSrc = config.srcGraph;
    auto fileDst = config.dstGraph;
    if (config.mt == CONV_MTX_BEL)
    {
        mwriter.write_market_bel<uint, int>(fileSrc, fileDst, false);
        return;
    }

    if (config.mt == CONV_TSV_BEL)
    {
        mwriter.write_tsv_bel<uint64, uint64>(fileSrc, fileDst);
        return;
    }

    if (config.mt == CONV_TSV_MTX)
    {
        mwriter.write_tsv_market<uint, int>(fileSrc, fileDst);
        return;
    }

    if (config.mt == CONV_BEL_MTX)
    {
        mwriter.write_bel_market<uint, int>(fileSrc, fileDst);
        return;
    }

    if (config.mt == CONV_TXT_BEL)
    {
        mwriter.write_txt_bel<uint, uint>(fileSrc, fileDst, true, 2, 0);
        return;
    }

    Timer read_graph_timer;

    const char *matr = config.srcGraph;
    graph::EdgeListFile f(matr);
    std::vector<EdgeTy<uint>> edges;
    std::vector<EdgeTy<uint>> fileEdges;
    auto lowerTriangular = [](const Edge &e)
    { return e.first > e.second; };
    auto upperTriangular = [](const Edge &e)
    { return e.first < e.second; };
    auto full = [](const Edge &e)
    { return false; };

    while (f.get_edges(fileEdges, 100))
    {
        edges.insert(edges.end(), fileEdges.begin(), fileEdges.end());
    }

    if (config.sortEdges)
    {
        f.sort_edges(edges);
    }

    graph::CSRCOO<uint> csrcoo;
    if (config.orient == Upper)
        csrcoo = graph::CSRCOO<uint>::from_edgelist(edges, lowerTriangular);
    else if (config.orient == Lower)
        csrcoo = graph::CSRCOO<uint>::from_edgelist(edges, upperTriangular);
    else
        csrcoo = graph::CSRCOO<uint>::from_edgelist(edges, full);

    uint n = csrcoo.num_rows();
    uint m = csrcoo.nnz();
    Log(debug, "value of n: %u\n", n);
    Log(debug, "value of m: %u\n", m);

    graph::COOCSRGraph<uint> g;
    g.capacity = m;
    g.numEdges = m;
    g.numNodes = n;

    g.rowPtr = new graph::GPUArray<uint>("Row pointer", AllocationTypeEnum::gpu, n + 1, config.deviceId, true);
    g.rowInd = new graph::GPUArray<uint>("Src Index", AllocationTypeEnum::gpu, m, config.deviceId, true);
    g.colInd = new graph::GPUArray<uint>("Dst Index", AllocationTypeEnum::gpu, m, config.deviceId, true);

    uint *rp, *ri, *ci;
    hipHostMalloc((void **)&rp, (n + 1) * sizeof(uint));
    hipHostMalloc((void **)&ri, (m) * sizeof(uint));
    hipHostMalloc((void **)&ci, (m) * sizeof(uint));

    CUDA_RUNTIME(hipMemcpy(rp, csrcoo.row_ptr(), (n + 1) * sizeof(uint), hipMemcpyKind::hipMemcpyHostToHost));
    CUDA_RUNTIME(hipMemcpy(ri, csrcoo.row_ind(), (m) * sizeof(uint), hipMemcpyKind::hipMemcpyHostToHost));
    CUDA_RUNTIME(hipMemcpy(ci, csrcoo.col_ind(), (m) * sizeof(uint), hipMemcpyKind::hipMemcpyHostToHost));

    g.rowPtr->cdata() = rp;
    g.rowInd->cdata() = ri;
    g.colInd->cdata() = ci;

    Log(info, "Read graph time: %f s", read_graph_timer.elapsed());

    uint dv = 32;
    typedef unsigned int ttt;
    config.cutoff = get_stats(m, n, n, g.rowPtr->cdata(), g.colInd->cdata());
    Log(debug, "cutoff: %u", config.cutoff);

    if (config.printStats)
    {
        MatrixStats(m, n, n, g.rowPtr->cdata(), g.colInd->cdata());
        PrintMtarixStruct(m, n, n, g.rowPtr->cdata(), g.colInd->cdata());

        ////////////////// intersection !!
        printf("Now # of bytes we need to make this matrix binary encoded !!\n");

        uint64 sum = 0;
        uint64 sumc = 0;
        for (uint i = 0; i < n; i++)
        {
            uint s = g.rowPtr->cdata()[i];
            uint d = g.rowPtr->cdata()[i + 1];
            uint deg = d - s;

            // if(i >=37 && i<44)
            // {
            // 	printf("For %u, %u, %u, %u\n", i, d-s, g.colInd->cdata()[s], g.colInd->cdata()[s + 1]);
            // }

            // if (deg > 128)
            {
                uint64 v = deg * (deg + dv - 1) / dv;
                sum += v;

                // now the compressed one :D
                uint64 nelem8 = deg / dv;
                uint64 rem = deg - nelem8 * dv;

                sumc += dv * nelem8 * (1 + nelem8) / 2;
                sumc += rem * (1 + nelem8);
            }
        }

        printf("n = %u, m = %u, elements = %llu\n", n, m, sum);
        printf("n = %u, m = %u, elements = %llu\n", n, m, sumc);

        uint src = 3541; // index id
        uint s = g.rowPtr->cdata()[src];
        uint d = g.rowPtr->cdata()[src + 1];
        uint degree = d - s;
        while (degree < 50)
        {
            src++;
            s = g.rowPtr->cdata()[src];
            d = g.rowPtr->cdata()[src + 1];
            degree = d - s;
        }

        uint divisions = (degree + dv - 1) / dv;
        graph::GPUArray<ttt> node_be("BE", unified, divisions * degree, 0);
        node_be.setAll(0, true);
        for (uint i = 0; i < degree; i++)
        {
            uint dst = g.colInd->cdata()[i + s];
            uint dstStart = g.rowPtr->cdata()[dst];
            uint dstEnd = g.rowPtr->cdata()[dst + 1];
            uint dstDegree = dstEnd - dstStart;

            // Intersect Src, Dst
            uint s1 = 0, s2 = 0;
            bool loadA = true, loadB = true;
            uint a, b;
            uint rsi = 0;
            uint offset = 0;
            while (s1 < degree && s2 < dstDegree)
            {

                if (loadA)
                {
                    a = g.colInd->cdata()[s1 + s];
                    loadA = false;
                }
                if (loadB)
                {
                    b = g.colInd->cdata()[s2 + dstStart];
                    loadB = false;
                }

                if (a == b)
                {
                    uint startIndex = i * divisions;
                    uint divIndex = s1 / dv;
                    uint inDivIndex = s1 % dv;
                    node_be.cdata()[startIndex + divIndex] |= (1 << inDivIndex);

                    // i and s1
                    // if (i > 0)
                    //{
                    //	if (i > s1)
                    //	{
                    //		uint ss = i / dv;
                    //		uint sum = dv * ss * (ss + 1) / 2;
                    //		uint sr = i % dv;
                    //		uint sumr = sr * ((i + dv - 1) / dv) - 1;

                    //		rsi = sum + sumr;
                    //		offset = s1 / dv;
                    //		uint numBytes = (i + dv - 1) / dv;
                    //		uint byteIndex = s1 % dv;

                    //		//Encode
                    //		node_be.cdata()[rsi + offset] |= (1 << byteIndex);

                    //	}
                    //	else
                    //	{
                    //		uint ss = s1 / dv;
                    //		uint sum = dv * ss * (ss + 1) / 2;
                    //		uint sr = s1 % dv;
                    //		uint sumr = sr * ((s1 + dv - 1) / dv) - 1;

                    //		rsi = sum + sumr;
                    //		offset = i / dv;
                    //		uint numBytes = (s1 + dv - 1) / dv;
                    //		uint byteIndex = i % dv;
                    //		node_be.cdata()[rsi + offset] |= (1 << byteIndex);
                    //	}

                    ++s1;
                    ++s2;
                    loadA = true;
                    loadB = true;
                }
                else if (a < b)
                {
                    ++s1;
                    loadA = true;
                }
                else
                {
                    ++s2;
                    loadB = true;
                }
            }
        }
    }

    Timer total_timer;

    graph::COOCSRGraph_d<uint> *gd = (graph::COOCSRGraph_d<uint> *)malloc(sizeof(graph::COOCSRGraph_d<uint>));

    gd->numNodes = g.numNodes;
    gd->numEdges = g.numEdges;
    gd->capacity = g.capacity;

    size_t mf, ma;
    g.rowPtr->switch_to_gpu(config.deviceId, g.numNodes + 1);
    hipDeviceSynchronize();
    Log(debug, "Moved rowPtr to device memory");
    hipMemGetInfo(&mf, &ma);
    // std::cout << "free: " << mf << " total: " << ma << std::endl;
    gd->rowPtr = g.rowPtr->gdata();

    g.rowInd->switch_to_gpu(config.deviceId, g.numEdges);
    hipDeviceSynchronize();
    Log(debug, "Moved rowIndices to device memory");
    hipMemGetInfo(&mf, &ma);
    // std::cout << "free: " << mf << " total: " << ma << std::endl;)
    g.colInd->switch_to_gpu(config.deviceId, g.numEdges);
    hipDeviceSynchronize();
    Log(debug, "Moved colIndices to device memory");
    hipMemGetInfo(&mf, &ma);
    // std::cout << "free: " << mf << " total: " << ma << std::endl;
    gd->rowInd = g.rowInd->gdata();
    gd->colInd = g.colInd->gdata();

    Log(debug, "Moved graph to device memory");
    hipHostFree(rp);
    hipHostFree(ri);
    hipHostFree(ci);

    Log(info, "Transfer Time: %f s", total_timer.elapsed());

    if (config.mt == GRAPH_MATCH || config.mt == GRAPH_COUNT)
    {
        // Read Template graph from file
        graph::EdgeListFile patFile(config.patGraph);
        std::vector<EdgeTy<uint>> patEdges;
        std::vector<EdgeTy<uint>> patFileEdges;

        while (patFile.get_edges(patFileEdges, 10 * 10))
        {
            patEdges.insert(patEdges.end(), patFileEdges.begin(), patFileEdges.end());
        }

        graph::CSRCOO<uint> patCsrcoo = graph::CSRCOO<uint>::from_edgelist(patEdges);
        graph::COOCSRGraph<uint> patG;
        patG.capacity = patCsrcoo.nnz();
        patG.numEdges = patCsrcoo.nnz();
        patG.numNodes = patCsrcoo.num_rows();

        patG.rowPtr = new graph::GPUArray<uint>("Row pointer", AllocationTypeEnum::cpuonly);
        patG.rowInd = new graph::GPUArray<uint>("Src index", AllocationTypeEnum::cpuonly);
        patG.colInd = new graph::GPUArray<uint>("Dst index", AllocationTypeEnum::cpuonly);

        patG.rowPtr->cdata() = patCsrcoo.row_ptr();
        patG.rowInd->cdata() = patCsrcoo.row_ind();
        patG.colInd->cdata() = patCsrcoo.col_ind();

        graph::SG_Match<uint> *sgm = new graph::SG_Match<uint>(config.mt, config.processBy, config.deviceId, config.cutoff, config.ndev);
        sgm->run(*gd, patG);

        // Clean up
        delete sgm;
    }

    return 0;
}
